#include "hip/hip_runtime.h"
#include "recommendations_kernel.h"

void allocateDeviceMemory(void ** d_data, size_t size);
void copyToDeviceMemory(void * d_data, void * h_data, size_t size);
void copyFromDeviceMemory(void * h_data, void * d_data, size_t size);

//basic kernel function to compute UU similarity from csr rating data
__global__ void csrSimilarityKernel(unsigned int dim, unsigned int * csrRowPtr, 
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output) {

    // get row ids for which dot product needs to be computed 
    unsigned int row_x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int row_y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((row_x >= dim) || (row_y >= dim))
        // out of bounds
        return;
    if (row_x == row_y){
        output[row_y + row_x * dim] = 0.0f;
        return;
    }
    float row_x_norm = userEuclideanNorm[row_x];
    float row_y_norm = userEuclideanNorm[row_y];
    if(!row_x_norm || !row_y_norm) {
        // if either is a zero vector
        output[row_y + row_x * dim] = 0.0f;
        return;
    }

    unsigned int id_x = csrRowPtr[row_x]; // set to start of row 1
    unsigned int id_y = csrRowPtr[row_y]; // set to start of row 2
                                
    unsigned int col_id_x, col_id_y;
    // compute similarity
    float similarity = 0.0f;
    while (id_x < csrRowPtr[row_x + 1] && id_y < csrRowPtr[row_y + 1]) {
        col_id_x = csrColIdx[id_x];
        col_id_y = csrColIdx[id_y];
        if (col_id_x == col_id_y) {
            similarity += csrData[id_x] * csrData[id_y];
            id_x ++;
            id_y ++;
        }
        else if (col_id_x < col_id_y) {
            id_x ++;
        }
        else {
            id_y ++;  
        }
    }
    output[row_y + dim * row_x] = similarity/(row_x_norm * row_y_norm);
}

// coarsened kernel where one thread writes to two result locations.
__global__ void csrSimilarityKernelCoarsened(unsigned int dim, unsigned int * csrRowPtr,
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output){
    unsigned int t_id = threadIdx.x + blockIdx.x * blockDim.x;
    // check out of bounds
    if(t_id >= ((dim-1)*dim)/2)
        return;
    // determine row ids for which dot product needs to be computed 
    unsigned int row_x = 0;
    unsigned int row_y = 0;
    unsigned int subtractor = dim - 1;
    while(t_id >= subtractor) {
        row_x++;
        t_id -= subtractor;
        subtractor--;
    }
    row_y = row_x + t_id + 1;
    float row_x_norm = userEuclideanNorm[row_x];
    float row_y_norm = userEuclideanNorm[row_y];
    // if either is a zero vector
    if(!row_x_norm || !row_y_norm) {
        output[row_y + row_x * dim] = 0.0f;
        return;
    }
    unsigned int id_x = csrRowPtr[row_x]; // set to start of row 1
    unsigned int id_y = csrRowPtr[row_y]; // set to start of row 2
                                
    unsigned int col_id_x, col_id_y;
    // compute similarity
    float similarity = 0.0f;
    while (id_x < csrRowPtr[row_x + 1] && id_y < csrRowPtr[row_y + 1]) {
        col_id_x = csrColIdx[id_x];
        col_id_y = csrColIdx[id_y];
        if (col_id_x == col_id_y) {
            similarity += csrData[id_x] * csrData[id_y];
            id_x ++;
            id_y ++;
        }
        else if (col_id_x < col_id_y) {
            id_x ++;
        }
        else {
            id_y ++;  
        }
    }
    output[row_y + dim * row_x] = similarity/(row_x_norm * row_y_norm);
    output[row_x + dim * row_y] = similarity/(row_x_norm * row_y_norm);
}

__global__ void csrSimilarityKernelShared(unsigned int dim, unsigned int * csrRowPtr,
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output) {

    // row info
    __shared__ unsigned int row_start_x; // start index
    __shared__ unsigned int row_end_x; // end index 
    __shared__ float data_x[TILE_SIZE]; // row data 
    __shared__ unsigned int cols_x[TILE_SIZE];// col ids
    __shared__ float row_x_norm; // euclidean norm

    unsigned int tid = threadIdx.x;
    unsigned int row_x = blockIdx.x;
    // initialize row ptrs
    if (tid == 0) {
        row_start_x = csrRowPtr[row_x];
        row_end_x = csrRowPtr[row_x + 1];
        row_x_norm = userEuclideanNorm[row_x];  
    } 
    // make sure the basic row info is loaded
    __syncthreads(); 
    
    // load data into shared memory
    unsigned int tile_idx = tid;
    unsigned int csr_idx = tile_idx + row_start_x;
    while (csr_idx < row_end_x && tile_idx < TILE_SIZE) {
        data_x[tile_idx] = csrData[csr_idx];
        cols_x[tile_idx] = csrColIdx[csr_idx];
        tile_idx += blockDim.x;
        csr_idx += blockDim.x;
    }
    // make sure the row data and col ids are loaded
    __syncthreads();

    unsigned int row_y = tid + blockIdx.x + 1;
    unsigned int id_x;
    unsigned int end_x = row_end_x - row_start_x;
    unsigned int id_y;
    unsigned int end_y;
    float row_y_norm; 
    while (row_y < dim) {
       id_x = 0;
       id_y = csrRowPtr[row_y];
       end_y = csrRowPtr[row_y + 1];
       row_y_norm = userEuclideanNorm[row_y];
       if (!row_x_norm || !row_y_norm) {
           output[row_y + dim * row_x] = 0.0f;
           output[row_x + dim * row_y] = 0.0f;
           row_y += blockDim.x;
           continue;
       }
       
       unsigned int col_id_x, col_id_y;
       // compute similarity
       float similarity = 0.0f;
       while (id_x < end_x && id_y < end_y) {
           col_id_x = cols_x[id_x];
           col_id_y = csrColIdx[id_y];
           if (col_id_x == col_id_y) {
               similarity += data_x[id_x] * csrData[id_y];
               id_x ++;
               id_y ++;
           }
           else if (col_id_x < col_id_y) {
               id_x ++;
           }
           else {
               id_y ++;
           }
       }
       similarity /= (row_x_norm * row_y_norm);
       output[row_y + dim * row_x] = similarity;
       output[row_x + dim * row_y] = similarity;
       row_y += blockDim.x;
    } 
}


//wrapper to kernel function
SimilarityMatrix computeSimilarityParallel(RatingsMatrixCSR &ratingMatrix) {
   
    unsigned int dim = ratingMatrix.rowPtrs.size() - 1;
    SimilarityMatrix similarityMatrix = {nullptr, (unsigned int) dim};
    //device DS
    unsigned int *csrRowPtr_d;
    unsigned int *csrColIdx_d;
    float *csrData_d;
    float *userEuclideanNorm_d;
    float *output_d;    
 
    //allocate memory for row ptr
    allocateDeviceMemory((void **)&csrRowPtr_d, sizeof(unsigned int) * (dim + 1));
    //allocate memory for col ids
    allocateDeviceMemory((void **)&csrColIdx_d, sizeof(unsigned int) * ratingMatrix.cols.size());
    //allocate memory for normalized ratings data
    allocateDeviceMemory((void **)&csrData_d, sizeof(float) * ratingMatrix.data.size());
    //allocate memory for user euclidien distance
    allocateDeviceMemory((void **)&userEuclideanNorm_d, sizeof(float) * ratingMatrix.userEuclideanNorm.size());
    //allocate memory for output
    allocateDeviceMemory((void **)&output_d, sizeof(float) * (dim * dim));   

    //copy row ptr to Device Memory
    unsigned int * csrRowPtr = &ratingMatrix.rowPtrs[0];
    copyToDeviceMemory(csrRowPtr_d, csrRowPtr, sizeof(unsigned int) * (dim + 1));
    //copy cold ids to Device Memory
    unsigned int * csrColIdx =  &ratingMatrix.cols[0];
    copyToDeviceMemory(csrColIdx_d, csrColIdx, sizeof(unsigned int) * ratingMatrix.cols.size());
    //copy data to Device Memory
    float * csrData = &ratingMatrix.data[0];
    copyToDeviceMemory(csrData_d, csrData, sizeof(float) * ratingMatrix.data.size());
    //copy euclidean norm to device memory
    float * userEuclideanNorm = &ratingMatrix.userEuclideanNorm[0];
    copyToDeviceMemory(userEuclideanNorm_d, userEuclideanNorm, sizeof(float) * ratingMatrix.userEuclideanNorm.size());    

    //allocate memory for similarities in host
    similarityMatrix.similarities = (float *) malloc(sizeof(float) * (dim * dim));

    //call csr kernel 1
    dim3 grid_dim, block_dim;
    block_dim.x = BLOCK_DIM_X;
    block_dim.y = BLOCK_DIM_Y;
    grid_dim.x = ceil((float)dim/BLOCK_DIM_X);
    grid_dim.y = ceil((float)dim/BLOCK_DIM_Y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f; 
  
    ////BASIC KERNEL////
 
    // hipEventRecord(start);
    // csrSimilarityKernel<<<grid_dim, block_dim>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    // hipEventRecord(stop);

    // //display results of kernel 1
    // copyFromDeviceMemory(similarityMatrix.similarities, output_d, sizeof(float) * (dim * dim));
    // hipEventSynchronize(stop);

    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Basic Kernel time: %f ms \n",  milliseconds);

    ////COARSENED KERNEL////

    // unsigned int bestThreadCount = ((dim-1)*dim)/2;
    // hipEventRecord(start);
    // csrSimilarityKernelCoarsened<<<ceil(bestThreadCount/1024.0f), 1024>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    // hipEventRecord(stop);

    // //display results of kernel 1
    // copyFromDeviceMemory(similarityMatrix.similarities, output_d, sizeof(float) * (dim * dim));
    // hipEventSynchronize(stop);

    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Coarsened Kernel time: %f ms \n",  milliseconds);
    
    ////SHARED KERNEL////

    hipEventRecord(start);
    csrSimilarityKernelShared<<<dim, BLOCK_SIZE>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    hipEventRecord(stop);

    //display results of kernel 2
    copyFromDeviceMemory(similarityMatrix.similarities, output_d, sizeof(float) * (dim * dim));
    hipEventSynchronize(stop);


    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Shared Kernel time: %f ms \n", milliseconds);

    //hipFree
    hipFree(csrRowPtr_d); 
    hipFree(csrColIdx_d);
    hipFree(csrData_d);
    hipFree(userEuclideanNorm_d);
    hipFree(output_d);

    return similarityMatrix;
}

void allocateDeviceMemory(void ** d_data, size_t size) 
{
    hipError_t cuda_ret;
    cuda_ret = hipMalloc(d_data, size);
    if(cuda_ret != hipSuccess) 
        printf("Unable to allocate device memory");    
}

void copyToDeviceMemory(void * d_data, void * h_data, size_t size) 
{
    hipError_t cuda_ret;
    cuda_ret = hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) 
        fprintf(stderr, "Unable to copy to device memory");
}

void copyFromDeviceMemory(void * h_data, void * d_data, size_t size)
{
    hipError_t cuda_ret;
    cuda_ret = hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess)
        fprintf(stderr, "Unable to copy from device memory");
}

