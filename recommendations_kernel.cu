#include "hip/hip_runtime.h"
#include "recommendations_kernel.h"

#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32
#define BLOCK_SIZE 32
#define TILE_SIZE 256 //assumption max TILE_SIZE ratings per row

void allocateDeviceMemory(void ** d_data, size_t size);
void copyToDeviceMemory(void * d_data, void * h_data, size_t size);
void copyFromDeviceMemory(void * h_data, void * d_data, size_t size);

//basic kernel function to compute UU similarity from csr rating data
__global__ void csrSimilarityKernel(unsigned int dim, unsigned int * csrRowPtr, 
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output) {

    //get row ids to which dot product needs to be computed 
    unsigned int row_x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int row_y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((row_x < dim) && (row_y < dim)) { 
        float row_x_norm = userEuclideanNorm[row_x];
        float row_y_norm = userEuclideanNorm[row_y]; 
        //if any of the rows are all zeros
        if (!row_x_norm || !row_y_norm) {
            output[row_y + dim * row_x] = 0.0f; //similarity is 0
            return;
        }

        unsigned int id_x = csrRowPtr[row_x]; //start of row 1
        unsigned int end_x = csrRowPtr[row_x + 1];//end of row 1 
        unsigned int id_y = csrRowPtr[row_y]; //start of row 2
        unsigned int end_y = csrRowPtr[row_y + 1];//end of row 2 
                                    
        unsigned int col_id_x, col_id_y;
        //compute similarity
        float similarity = 0.0f;
        while (id_x < end_x && id_y < end_y) {
            col_id_x = csrColIdx[id_x];
            col_id_y = csrColIdx[id_y];
            if (col_id_x == col_id_y) {
                similarity += csrData[id_x] * csrData[id_y];
                id_x ++;
                id_y ++;
            }
            else if (col_id_x < col_id_y) {
                id_x ++;
            }
            else {
                id_y ++;  
            }
        }
        output[row_y + dim * row_x] = similarity/(row_x_norm * row_y_norm);
    }  
}


//test kernel 2
__global__ void csrSimilarityKernelShared(unsigned int dim, unsigned int * csrRowPtr,
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output) {

    //row info 
    __shared__ unsigned int row_start_x;//start index
    __shared__ unsigned int row_end_x;//end index 
    __shared__ float data_x[TILE_SIZE];//row data 
    __shared__ unsigned int cols_x[TILE_SIZE];//col ids
    __shared__ float row_x_norm;//euclidean norm

    unsigned int tid = threadIdx.x;
    unsigned int row_x = blockIdx.x;
    //initialize row ptrs
    if (tid == 0) {
        row_start_x = csrRowPtr[row_x];
        row_end_x = csrRowPtr[row_x + 1];
        row_x_norm = userEuclideanNorm[row_x];  
    } 
    //make sure the basic row info is loaded
    __syncthreads(); 
    
    //load data into shared memory
    unsigned int tile_idx = tid;
    unsigned int csr_idx = tile_idx + row_start_x;
    while (csr_idx < row_end_x && tile_idx < TILE_SIZE) {
        data_x[tile_idx] = csrData[csr_idx];
        cols_x[tile_idx] = csrColIdx[csr_idx];
        tile_idx += blockDim.x; 
        csr_idx += blockDim.x; 
    }
    //make sure the row data and col ids are loaded
    __syncthreads();

    unsigned int row_y = tid + blockIdx.x;
    unsigned int id_x;
    unsigned int end_x = row_end_x - row_start_x;
    unsigned int id_y;
    unsigned int end_y;
    float row_y_norm; 
    while (row_y < dim) {
       id_x = 0;
       id_y = csrRowPtr[row_y];
       end_y = csrRowPtr[row_y + 1];
       row_y_norm = userEuclideanNorm[row_y];
       if (!row_x_norm || !row_y_norm) {
           output[row_y + dim * row_x] = 0.0f;
           output[row_x + dim * row_y] = 0.0f;
           row_y += blockDim.x;
           continue;
       }
       
       unsigned int col_id_x, col_id_y;
       //compute similarity
       float similarity = 0.0f;
       while (id_x < end_x && id_y < end_y) {
           col_id_x = cols_x[id_x];
           col_id_y = csrColIdx[id_y];
           if (col_id_x == col_id_y) {
               similarity += data_x[id_x] * csrData[id_y];
               id_x ++;
               id_y ++;
           }
           else if (col_id_x < col_id_y) {
               id_x ++;
           }
           else {
               id_y ++;
           }
       }
       similarity /= (row_x_norm * row_y_norm);
       output[row_y + dim * row_x] = similarity;
       output[row_x + dim * row_y] = similarity;
       row_y += blockDim.x;
    } 
}


//wrapper to kernel function
SimilarityMatrix computeSimilarityParallel(RatingsMatrixCSR &ratingMatrix) {
   
    unsigned int dim = ratingMatrix.rowPtrs.size() - 1;
    SimilarityMatrix similarityMatrix = {nullptr, (unsigned int) dim};
    //device DS
    unsigned int *csrRowPtr_d;
    unsigned int *csrColIdx_d;
    float *csrData_d;
    float *userEuclideanNorm_d;
    float *output_d;    
 
    //allocate memory for row ptr
    allocateDeviceMemory((void **)&csrRowPtr_d, sizeof(unsigned int) * (dim + 1));
    //allocate memory for col ids
    allocateDeviceMemory((void **)&csrColIdx_d, sizeof(unsigned int) * ratingMatrix.cols.size());
    //allocate memory for normalized ratings data
    allocateDeviceMemory((void **)&csrData_d, sizeof(float) * ratingMatrix.data.size());
    //allocate memory for user euclidien distance
    allocateDeviceMemory((void **)&userEuclideanNorm_d, sizeof(float) * ratingMatrix.userEuclideanNorm.size());
    //allocate memory for output
    allocateDeviceMemory((void **)&output_d, sizeof(float) * (dim * dim));   

    //copy row ptr to Device Memory
    unsigned int * csrRowPtr = &ratingMatrix.rowPtrs[0];
    copyToDeviceMemory(csrRowPtr_d, csrRowPtr, sizeof(unsigned int) * (dim + 1));
    //copy cold ids to Device Memory
    unsigned int * csrColIdx =  &ratingMatrix.cols[0];
    copyToDeviceMemory(csrColIdx_d, csrColIdx, sizeof(unsigned int) * ratingMatrix.cols.size());
    //copy data to Device Memory
    float * csrData = &ratingMatrix.data[0];
    copyToDeviceMemory(csrData_d, csrData, sizeof(float) * ratingMatrix.data.size());
    //copy euclidean norm to device memory
    float * userEuclideanNorm = &ratingMatrix.userEuclideanNorm[0];
    copyToDeviceMemory(userEuclideanNorm_d, userEuclideanNorm, sizeof(float) * ratingMatrix.userEuclideanNorm.size());    

    //allocate memory for similarities in host
    similarityMatrix.similarities = (float *) malloc(sizeof(float) * (dim * dim));

    //call csr kernel 1
    dim3 grid_dim, block_dim;
    block_dim.x = BLOCK_DIM_X;
    block_dim.y = BLOCK_DIM_Y;
    grid_dim.x = ceil((float)dim/BLOCK_DIM_X);
    grid_dim.y = ceil((float)dim/BLOCK_DIM_Y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f; 
  
    ////BASIC KERNEL////
 
    hipEventRecord(start);
    csrSimilarityKernel<<<grid_dim, block_dim>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    hipEventRecord(stop);
    
    //hipDeviceSynchronize();

    //display results of kernel 1
    copyFromDeviceMemory(similarityMatrix.similarities, output_d, sizeof(float) * (dim * dim));
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Basic Kernel time: %f ms \n",  milliseconds);
    
    ////SHARED KERNEL////

    /*hipEventRecord(start);
    csrSimilarityKernelShared<<<dim, BLOCK_SIZE>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    hipEventRecord(stop);    

    //hipDeviceSynchronize();
 
    //display results of kernel 2
    copyFromDeviceMemory(similarityMatrix.similarities, output_d, sizeof(float) * (dim * dim));     
    hipEventSynchronize(stop);
   

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Shared Kernel time: %f ms \n", milliseconds);*/

    //hipFree
    hipFree(csrRowPtr_d); 
    hipFree(csrColIdx_d);
    hipFree(csrData_d);
    hipFree(userEuclideanNorm_d);
    hipFree(output_d);
    
    //similarityMatrix.similarities = output; 
    return similarityMatrix;
}

void allocateDeviceMemory(void ** d_data, size_t size) 
{
    hipError_t cuda_ret;
    cuda_ret = hipMalloc(d_data, size);
    if(cuda_ret != hipSuccess) 
        printf("Unable to allocate device memory");    
}

void copyToDeviceMemory(void * d_data, void * h_data, size_t size) 
{
    hipError_t cuda_ret;
    cuda_ret = hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) 
        fprintf(stderr, "Unable to copy to device memory");
}

void copyFromDeviceMemory(void * h_data, void * d_data, size_t size)
{
    hipError_t cuda_ret;
    cuda_ret = hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess)
        fprintf(stderr, "Unable to copy from device memory");
}

