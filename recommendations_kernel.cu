#include "hip/hip_runtime.h"
#include "recommendations_kernel.h"
#include <queue>
#include <algorithm>

//TODO tuning
#define BLOCK_SIZE_REC 64
#define BLOCK_SIZE_DIV 64 

void allocateDeviceMemory(void ** d_data, size_t size);
void copyToDeviceMemory(void * d_data, void * h_data, size_t size);
void copyFromDeviceMemory(void * h_data, void * d_data, size_t size);

//basic kernel function to compute UU similarity from csr rating data
__global__ void csrSimilarityKernel(unsigned int dim, unsigned int * csrRowPtr, 
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output) {

    // get row ids for which dot product needs to be computed 
    unsigned int row_x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int row_y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((row_x >= dim) || (row_y >= dim))
        // out of bounds
        return;
    if (row_x == row_y){
        output[row_y + row_x * dim] = 0.0f;
        return;
    }
    float row_x_norm = userEuclideanNorm[row_x];
    float row_y_norm = userEuclideanNorm[row_y];
    if(!row_x_norm || !row_y_norm) {
        // if either is a zero vector
        output[row_y + row_x * dim] = 0.0f;
        return;
    }

    unsigned int id_x = csrRowPtr[row_x]; // set to start of row 1
    unsigned int id_y = csrRowPtr[row_y]; // set to start of row 2
                                
    unsigned int col_id_x, col_id_y;
    // compute similarity
    float similarity = 0.0f;
    while (id_x < csrRowPtr[row_x + 1] && id_y < csrRowPtr[row_y + 1]) {
        col_id_x = csrColIdx[id_x];
        col_id_y = csrColIdx[id_y];
        if (col_id_x == col_id_y) {
            similarity += csrData[id_x] * csrData[id_y];
            id_x ++;
            id_y ++;
        }
        else if (col_id_x < col_id_y) {
            id_x ++;
        }
        else {
            id_y ++;  
        }
    }
    output[row_y + dim * row_x] = similarity/(row_x_norm * row_y_norm);
}

// coarsened kernel where one thread writes to two result locations.
__global__ void csrSimilarityKernelCoarsened(unsigned int dim, unsigned int * csrRowPtr,
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output){
    unsigned int t_id = threadIdx.x + blockIdx.x * blockDim.x;
    // check out of bounds
    if(t_id >= ((dim-1)*dim)/2)
        return;
    // determine row ids for which dot product needs to be computed 
    unsigned int row_x = 0;
    unsigned int row_y = 0;
    unsigned int subtractor = dim - 1;
    while(t_id >= subtractor) {
        row_x++;
        t_id -= subtractor;
        subtractor--;
    }
    row_y = row_x + t_id + 1;
    float row_x_norm = userEuclideanNorm[row_x];
    float row_y_norm = userEuclideanNorm[row_y];
    // if either is a zero vector
    if(!row_x_norm || !row_y_norm) {
        output[row_y + row_x * dim] = 0.0f;
        return;
    }
    unsigned int id_x = csrRowPtr[row_x]; // set to start of row 1
    unsigned int id_y = csrRowPtr[row_y]; // set to start of row 2
                                
    unsigned int col_id_x, col_id_y;
    // compute similarity
    float similarity = 0.0f;
    while (id_x < csrRowPtr[row_x + 1] && id_y < csrRowPtr[row_y + 1]) {
        col_id_x = csrColIdx[id_x];
        col_id_y = csrColIdx[id_y];
        if (col_id_x == col_id_y) {
            similarity += csrData[id_x] * csrData[id_y];
            id_x ++;
            id_y ++;
        }
        else if (col_id_x < col_id_y) {
            id_x ++;
        }
        else {
            id_y ++;  
        }
    }
    output[row_y + dim * row_x] = similarity/(row_x_norm * row_y_norm);
    output[row_x + dim * row_y] = similarity/(row_x_norm * row_y_norm);
}

//shared implementation of csr similarity kernel
__global__ void csrSimilarityKernelShared(unsigned int dim, unsigned int * csrRowPtr,
                 unsigned int * csrColIdx, float * csrData, float * userEuclideanNorm, float * output) {

    // row info
    __shared__ unsigned int row_start_x_sh; // start index
    __shared__ unsigned int row_end_x_sh; // end index 
    __shared__ float data_x[TILE_SIZE]; // row data 
    __shared__ unsigned int cols_x[TILE_SIZE];// col ids
    __shared__ float row_x_norm_sh; //euclidean norm

    unsigned int tid = threadIdx.x;
    unsigned int row_x = blockIdx.x;
    
    // initialize row ptrs
    if (tid == 0) {
        row_start_x_sh = csrRowPtr[row_x];
        row_end_x_sh = csrRowPtr[row_x + 1];
        row_x_norm_sh = userEuclideanNorm[row_x];  
    } 
    // make sure the basic row info is loaded
    __syncthreads(); 
    
    unsigned int row_start_x = row_start_x_sh;
    unsigned int row_end_x = row_end_x_sh;
    unsigned int tile_idx = tid;
    unsigned int csr_idx = tile_idx + row_start_x;
    //load csr data into shared memory
    while (csr_idx < row_end_x && tile_idx < TILE_SIZE) {
        data_x[tile_idx] = csrData[csr_idx];
        cols_x[tile_idx] = csrColIdx[csr_idx];
        tile_idx += blockDim.x;
        csr_idx += blockDim.x;
    }
    // make sure the row data and col ids are loaded
    __syncthreads();

    
    unsigned int row_y = tid + blockIdx.x + 1;
    unsigned int id_x;
    unsigned int end_x = row_end_x - row_start_x;
    unsigned int id_y;
    unsigned int end_y;
    float row_x_norm = row_x_norm_sh;
    float row_y_norm; 
    while (row_y < dim) {
       id_x = 0;
       id_y = csrRowPtr[row_y];
       end_y = csrRowPtr[row_y + 1];
       row_y_norm = userEuclideanNorm[row_y];
       if (!row_x_norm || !row_y_norm) {
           output[row_y + dim * row_x] = 0.0f;
           output[row_x + dim * row_y] = 0.0f;
           row_y += blockDim.x;
           continue;
       }
       
       unsigned int col_id_x, col_id_y;
       // compute similarity
       float similarity = 0.0f;
       while (id_x < end_x && id_y < end_y) {
           col_id_x = cols_x[id_x];
           col_id_y = csrColIdx[id_y];
           if (col_id_x == col_id_y) {
               similarity += data_x[id_x] * csrData[id_y];
               id_x ++;
               id_y ++;
           }
           else if (col_id_x < col_id_y) {
               id_x ++;
           }
           else {
               id_y ++;
           }
       }
       similarity /= (row_x_norm * row_y_norm);
       output[row_y + dim * row_x] = similarity;
       output[row_x + dim * row_y] = similarity;
       row_y += blockDim.x;
    } 
}

//kernel to do vector division to compute final prediction score
__global__ void computeFinalPredictionScores(ItemRating *recommendations, float *similarities_sum,
                                            unsigned int rec_size, float userMean)  {
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float score;
    if (tid < rec_size) {
        score = recommendations[tid].rating;
        float similarity_sum = similarities_sum[tid]; 
        if (similarity_sum > 0.0f) {
            score /= similarity_sum;
            score += userMean;
            recommendations[tid].rating = score;
        }
    }
}


//fetches the item index if it exists
__device__ int getItemIndex(unsigned int * col_ids, unsigned int count, unsigned int item_id) {
    //simple search
    unsigned int col;
    for (int i = 0; i < count; i++) {
        col = col_ids[i];
        if (item_id == col) {
            return i;
        }
        else if (item_id < col) {
            return -1;
        }
    }
    return -1;
}


//kernel to compute predictions for user
__global__ void computePredictionsForUserKernel(unsigned int *csrRowPtr, unsigned int *csrColIdx, float *csrData,
                                                ItemRating *recommendations, Similarity *sortedNeighbours, float *similarities_sum,
                                                unsigned int rec_size, unsigned int neighbour_size) {
     //rating info of neighbour user
    __shared__ unsigned int row_start_sh;//start index
    __shared__ unsigned int row_end_sh;//end index
    __shared__ unsigned int row_count_sh;//total elements
    __shared__ float data[TILE_SIZE];//row data
    __shared__ unsigned int cols[TILE_SIZE];//col ids
    __shared__ Similarity neighbour;

    unsigned int tid = threadIdx.x;

    //load initial row info
    if (tid == 0) {
        neighbour = sortedNeighbours[blockIdx.x];
        unsigned int row = neighbour.userId;
        row_start_sh = csrRowPtr[row];
        row_end_sh = csrRowPtr[row + 1];
        //restricting data to TILE size  
        row_count_sh = fminf(TILE_SIZE, row_end_sh - row_start_sh);
    }
    __syncthreads();

    //load from Shared Memory to local regs
    unsigned int row_start = row_start_sh;
    unsigned int row_end = row_end_sh;
    unsigned int row_count = row_count_sh;

    unsigned int tile_idx = tid;
    unsigned int csr_idx = tile_idx + row_start;

    //load user data into shared memory
    while (csr_idx < row_end && tile_idx < TILE_SIZE) {
        data[tile_idx] = csrData[csr_idx];
        cols[tile_idx] = csrColIdx[csr_idx];
        tile_idx += blockDim.x;
        csr_idx += blockDim.x;
    }
    __syncthreads();

    unsigned int max_item_id = cols[row_count - 1];
    ItemRating item_rating;
    unsigned int item_itr = tid;
    int item_idx;
    float similarity = neighbour.similarityValue;
    float result;
    //iterate through input set of items
    while (item_itr < rec_size) {
        item_rating = recommendations[item_itr];
        //if item id exceeds maximum rated item id then return
        if (item_rating.item >  max_item_id) {
            return;
        }
        //check if item exists in user rated items (col ids)
        if ((item_idx = getItemIndex(cols, row_count, item_rating.item)) != -1) {
            result = data[item_idx] * similarity;
            //atomic add result score (numerator)
            atomicAdd(&(recommendations[item_itr].rating), result);
            //atomic add similarity sum (denominator)
            atomicAdd(&(similarities_sum[item_itr]), similarity);
        }
        item_itr += blockDim.x;
    }
}

//wrapper function to top n recs kernel
vector<ItemRating> calculateTopNRecommendationsForUserParallel(unsigned int *csrRowPtr_d, unsigned int *csrColIdx_d, float *csrData_d,
                                                 SimilarityMatrix similarityMatrix, vector<unsigned int> movieIds,
                                                 RatingsMatrixCSR &ratingsMatrix, unsigned int userId, unsigned int N) {

    //find the unrated items for this user (same as Gold)
    vector<ItemRating> recommendations;
    unsigned int item = ratingsMatrix.rowPtrs[userId];
    unsigned int end = ratingsMatrix.rowPtrs[userId + 1];
    for (auto &movieId : movieIds) {
        if (item >= end || movieId < ratingsMatrix.cols[item])
            recommendations.push_back(ItemRating{movieId, 0});
        else if (movieId == ratingsMatrix.cols[item])
            item += 1;
    }

    //add similar users into a priority queue
    priority_queue <Similarity, vector<Similarity>, greater<Similarity> > similarUsers;
    unsigned int neighbourhood_size = similarityMatrix.size/20; //considering 5% more similar users in neighbourhood
    for (unsigned int i = 0; i < similarityMatrix.size; i++) {
        
        float similarityValue = similarityMatrix.similarities[userId * similarityMatrix.size + i];
        //ignore any similarity that's not positive
        if (i == userId || similarityValue <= 0)
            continue;
        Similarity currUser = Similarity{i, similarityValue};
        if (similarUsers.size() < neighbourhood_size) {
            similarUsers.push(currUser);
        }
        else {
            if (currUser > similarUsers.top()) {
                similarUsers.pop();
                similarUsers.push(currUser);
            }
        }
    }

    ItemRating *recommendations_d;
    Similarity *similarUsers_d;
    float * similaritySum_d;

    //allocate memory
    allocateDeviceMemory((void **)&recommendations_d, sizeof(ItemRating) * recommendations.size());
    allocateDeviceMemory((void **)&similarUsers_d, sizeof(Similarity) * similarUsers.size());
    allocateDeviceMemory((void **)&similaritySum_d, sizeof(float) * recommendations.size());
    

    //copy to Device memory
    copyToDeviceMemory(recommendations_d, &recommendations[0], sizeof(ItemRating) * recommendations.size());
    copyToDeviceMemory(similarUsers_d, (Similarity *) &similarUsers.top(), sizeof(Similarity) * similarUsers.size());
    //initialize all values to 0
    hipMemset(similaritySum_d, 0, sizeof(float) * recommendations.size());

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f;     

    //the kernel starts here
    unsigned int noOfBlocks = similarUsers.size();
    hipEventRecord(start);
    computePredictionsForUserKernel<<<noOfBlocks, BLOCK_SIZE_REC>>>(csrRowPtr_d, csrColIdx_d, csrData_d, recommendations_d,
                                           similarUsers_d, similaritySum_d, recommendations.size(), similarUsers.size());
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Recommendations Kernel exec time: %f ms \n", milliseconds);    


    float userMean = ratingsMatrix.userMean[userId];
    noOfBlocks = ceil((float)recommendations.size()/BLOCK_SIZE_DIV);

    hipEventRecord(start);
    computeFinalPredictionScores<<<noOfBlocks, BLOCK_SIZE_DIV>>>(recommendations_d, similaritySum_d, recommendations.size(), userMean);
    hipEventRecord(stop);

    copyFromDeviceMemory(&recommendations[0], recommendations_d, sizeof(ItemRating) * recommendations.size());
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Division Kernel exec time: %f ms \n", milliseconds);


    //fetch top N recommendations
    priority_queue <ItemRating, vector<ItemRating>, greater<ItemRating> > topRecommendations;
    for (int i = 0; i < recommendations.size(); i++) {
        if (topRecommendations.size() < N) {
            topRecommendations.push(recommendations[i]);
        }
        else {
            if (recommendations[i] > topRecommendations.top()) {
                topRecommendations.pop();
                topRecommendations.push(recommendations[i]);
            }
        }
    }

    vector<ItemRating> sortedTopRecommendations;
    while (!topRecommendations.empty()) {
        sortedTopRecommendations.push_back(topRecommendations.top());
        topRecommendations.pop();        
    }

    sort(sortedTopRecommendations.begin(), sortedTopRecommendations.end(), greater<ItemRating>());
    return sortedTopRecommendations;
}


//wrapper to kernel function
SimilarityMatrix computeSimilarityParallel(unsigned int dim, unsigned int *csrRowPtr_d, unsigned int *csrColIdx_d,
                                      float *csrData_d,  float *userEuclideanNorm_d) {
      
    float *output_d;
    //allocate memory for output
    allocateDeviceMemory((void **)&output_d, sizeof(float) * (dim * dim));

    //call csr kernel 1
    dim3 grid_dim, block_dim;
    block_dim.x = BLOCK_DIM_X;
    block_dim.y = BLOCK_DIM_Y;
    grid_dim.x = ceil((float)dim/BLOCK_DIM_X);
    grid_dim.y = ceil((float)dim/BLOCK_DIM_Y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f; 

    float * similarities = (float *)malloc(sizeof(float) * (dim * dim));
  
    ////BASIC KERNEL////
 
    hipEventRecord(start);
    csrSimilarityKernel<<<grid_dim, block_dim>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    hipEventRecord(stop);

    // //display results of kernel 1
    copyFromDeviceMemory(similarities, output_d, sizeof(float) * (dim * dim));
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Basic Kernel time: %f ms \n",  milliseconds);

    ////COARSENED KERNEL////

    // unsigned int bestThreadCount = ((dim-1)*dim)/2;
    // hipEventRecord(start);
    // csrSimilarityKernelCoarsened<<<ceil(bestThreadCount/1024.0f), 1024>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    // hipEventRecord(stop);

    // //display results of kernel 1
    // copyFromDeviceMemory(similarities, output_d, sizeof(float) * (dim * dim));
    // hipEventSynchronize(stop);

    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Coarsened Kernel time: %f ms \n",  milliseconds);
    
    ////SHARED KERNEL////

    //hipEventRecord(start);
    //csrSimilarityKernelShared<<<dim, BLOCK_SIZE>>>(dim, csrRowPtr_d, csrColIdx_d, csrData_d, userEuclideanNorm_d, output_d);
    //hipEventRecord(stop);

    //copyFromDeviceMemory(similarities, output_d, sizeof(float) * (dim * dim));
    //hipEventSynchronize(stop);

    //hipEventElapsedTime(&milliseconds, start, stop);
    //printf("Shared Kernel time: %f ms \n", milliseconds);

    SimilarityMatrix outputSimilarityMatrix = {similarities, dim};
    return outputSimilarityMatrix;
}

void allocateMemoryToDevicePtrs(unsigned int dim, unsigned int **csrRowPtr_d, unsigned int **csrColIdx_d,
                                float **csrData_d, float **userEuclideanNorm_d,  RatingsMatrixCSR &ratingMatrix) {
    //allocate memory for row ptr
    allocateDeviceMemory((void **)csrRowPtr_d, sizeof(unsigned int) * (dim + 1));
    //allocate memory for col ids
    allocateDeviceMemory((void **)csrColIdx_d, sizeof(unsigned int) * ratingMatrix.cols.size());
    //allocate memory for normalized ratings data
    allocateDeviceMemory((void **)csrData_d, sizeof(float) * ratingMatrix.data.size());
    //allocate memory for user euclidien distance
    allocateDeviceMemory((void **)userEuclideanNorm_d, sizeof(float) * ratingMatrix.userEuclideanNorm.size());
}

void copyRatingsMatrixToDevicePtrs(unsigned int dim, unsigned int *csrRowPtr_d, unsigned int *csrColIdx_d,
                                   float *csrData_d, float *userEuclideanNorm_d, RatingsMatrixCSR &ratingMatrix) {

    //copy row ptr to Device Memory
    unsigned int * csrRowPtr = &ratingMatrix.rowPtrs[0];
    copyToDeviceMemory(csrRowPtr_d, csrRowPtr, sizeof(unsigned int) * (dim + 1));
    //copy cold ids to Device Memory
    unsigned int * csrColIdx =  &ratingMatrix.cols[0];
    copyToDeviceMemory(csrColIdx_d, csrColIdx, sizeof(unsigned int) * ratingMatrix.cols.size());
    //copy data to Device Memory
    float * csrData = &ratingMatrix.data[0];
    copyToDeviceMemory(csrData_d, csrData, sizeof(float) * ratingMatrix.data.size());
    //copy euclidean norm to device memory
    float * userEuclideanNorm = &ratingMatrix.userEuclideanNorm[0];
    copyToDeviceMemory(userEuclideanNorm_d, userEuclideanNorm, sizeof(float) * ratingMatrix.userEuclideanNorm.size());
}


void allocateDeviceMemory(void ** d_data, size_t size) 
{
    hipError_t cuda_ret;
    cuda_ret = hipMalloc(d_data, size);
    if(cuda_ret != hipSuccess) 
        printf("Unable to allocate device memory");    
}

void copyToDeviceMemory(void * d_data, void * h_data, size_t size) 
{
    hipError_t cuda_ret;
    cuda_ret = hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) 
        fprintf(stderr, "Unable to copy to device memory");
}

void copyFromDeviceMemory(void * h_data, void * d_data, size_t size)
{
    hipError_t cuda_ret;
    cuda_ret = hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess)
        fprintf(stderr, "Unable to copy from device memory");
}

